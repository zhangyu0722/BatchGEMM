#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <hipblas.h>
#include "../include/util.h"
#include "kernel.h"

#define N_RUNS 10

int  main (int argc, char** argv) {

	ErrChk(hipSetDevice(0));

	if(argc<2){
		printf("Usage: input the batch size\n");
		exit(EXIT_FAILURE);
	}

	int BATCH = atoi(argv[1]);
	//int TLP_thres = atoi(argv[2]);
	int TLP_thres = 65536*2;
	
	int *M;
	int *N;
	int *K;

	M = (int*) malloc(BATCH * sizeof(int));
	N = (int*) malloc(BATCH * sizeof(int));
	K = (int*) malloc(BATCH * sizeof(int));

	std::fstream fs;
	fs.open("../data/data_MN_K_512_128");
	if (!fs.is_open()){
		printf("Error opening input\n");
		exit(EXIT_FAILURE);
	}
	
	//read matrix config	
	for (int i=0; i<BATCH; ++i){
		fs>>M[i]>>N[i]>>K[i];
	}

    float **A;
	float **B;
	float **C;

	A = (float**) malloc(BATCH * sizeof(float*));
	B = (float**) malloc(BATCH * sizeof(float*));
	C = (float**) malloc(BATCH * sizeof(float*));

	for (int i=0; i<BATCH; ++i){
		ErrChk(hipMalloc((void**)&A[i], M[i]*K[i]*sizeof(float)));
		ErrChk(hipMalloc((void**)&B[i], K[i]*N[i]*sizeof(float)));
		ErrChk(hipMalloc((void**)&C[i], M[i]*N[i]*sizeof(float)));
	}

	float **dev_A;
	float **dev_B;
	float **dev_C;

    ErrChk(hipMalloc((void**)&dev_A, BATCH*sizeof(float*)));
    ErrChk(hipMalloc((void**)&dev_B, BATCH*sizeof(float*)));
    ErrChk(hipMalloc((void**)&dev_C, BATCH*sizeof(float*)));

	ErrChk(hipMemcpy(dev_A, A, BATCH*sizeof(float*), hipMemcpyHostToDevice));
	ErrChk(hipMemcpy(dev_B, B, BATCH*sizeof(float*), hipMemcpyHostToDevice));
	ErrChk(hipMemcpy(dev_C, C, BATCH*sizeof(float*), hipMemcpyHostToDevice));


	int *dev_M, *dev_N, *dev_K;
	ErrChk(hipMalloc((void**)&dev_M, BATCH*sizeof(int)));
	ErrChk(hipMalloc((void**)&dev_N, BATCH*sizeof(int)));
	ErrChk(hipMalloc((void**)&dev_K, BATCH*sizeof(int)));

	ErrChk(hipMemcpy(dev_M, M, BATCH*sizeof(int), hipMemcpyHostToDevice));
	ErrChk(hipMemcpy(dev_N, N, BATCH*sizeof(int), hipMemcpyHostToDevice));
	ErrChk(hipMemcpy(dev_K, K, BATCH*sizeof(int), hipMemcpyHostToDevice));

	
	float elapsedTime = 0.f;
    double time=0.f;
	float gflops_per_sec = 0.f;
	double gflops = 0.f;
	for (int i=0; i<BATCH; ++i)
		gflops += ((2 * int64_t(M[i]) * int64_t(N[i]) * int64_t(K[i])) + (2 * int64_t(M[i]) * int64_t(N[i])) ) / 1.0e9;
	hipEvent_t start, stop;

	//Tiling Strategy
	int TLP = 0;

	const int tile_size[6][2] = {
		16, 16,
		32, 32,
		64, 64,
		128, 64,
		64, 128,
		128, 128
	};
	
	int *t_strategy;
	t_strategy = (int*) malloc(BATCH * sizeof(int));

	int t;	
	for (t=0; t<6; ++t){
		TLP = 0;
		for (int j=0; j<BATCH; ++j)
			TLP += (M[j]/tile_size[t][0])*(N[j]/tile_size[t][1])*256;
		
		if (TLP < TLP_thres)
			break;
	}

	for (int j=0; j<BATCH; ++j){
	
		t_strategy[j] = 0;
		t = (t==6?5:t);

		if (tile_size[t][0] <= M[j] && tile_size[t][1] <= N[j])
			t_strategy[j] = t;
		else{
			for (int k=0; k<t; ++k){
				if (tile_size[k][0] == M[j] && tile_size[k][1] <= N[j]){
					t_strategy[j] = k;
				}
			}
		}
	}

	
	
	int *dev_T;
	ErrChk(hipMalloc((void**)&dev_T, BATCH*sizeof(int)));
	ErrChk(hipMemcpy(dev_T, t_strategy, BATCH*sizeof(int), hipMemcpyHostToDevice));

/*	
	//print the obtained tiling strategy
	for (int j=0; j<BATCH; ++j)
		printf("%d ", t_strategy[j]);
	printf("\n");
*/

	
	

	//Batching Strategy
	int *b_strategy;
	b_strategy = (int*) malloc(BATCH * sizeof(int));

	for (int j=0; j<BATCH; ++j){
		b_strategy[j] = 1;
	}

	for (int j=0; j<BATCH; ++j){
		TLP -= M[j]/2/tile_size[t_strategy[j]][0]*N[j]/tile_size[t_strategy[j]][1];
		if (TLP > TLP_thres && M[j]>t_strategy[j] && K[j]<=32)
			b_strategy[j] = 2;
	}


	int *dev_Ba;
	ErrChk(hipMalloc((void**)&dev_Ba, BATCH*sizeof(int)));
	ErrChk(hipMemcpy(dev_Ba, b_strategy, BATCH*sizeof(int), hipMemcpyHostToDevice));

	
/*
	//print the obtained batching strategy
	for (int j=0; j<BATCH; ++j)
		printf("%d ", b_strategy[j]);
	printf("\n");
*/


	


	//GEMM
    dim3 block_size;
    block_size.x = 256;
    block_size.y = 1;
	block_size.z = 1;

    dim3 grid_size;
	
    grid_size.x = M[0]/b_strategy[0]/tile_size[t_strategy[0]][0];
    grid_size.y = N[0]/b_strategy[0]/tile_size[t_strategy[0]][1];
	grid_size.z = BATCH;
	for (int j=1; j<BATCH; ++j){
		grid_size.x = (grid_size.x > M[j]/b_strategy[j]/tile_size[t_strategy[j]][0])? (grid_size.x):(M[j]/b_strategy[j]/tile_size[t_strategy[j]][0]);
		grid_size.y = (grid_size.y > N[j]/tile_size[t_strategy[j]][1])? (grid_size.y):(N[j]/tile_size[t_strategy[j]][1]);
	}

//	printf("%d %d %d\n", grid_size.x, grid_size.y, grid_size.z);

	//warm-up
	gemm_256<<<grid_size, block_size, sizeof(float)*4*128*8>>>(dev_M, dev_N, dev_K, dev_A, dev_B, dev_C, dev_T, dev_Ba);
	KernelErrChk();

	ErrChk(hipEventCreate(&start));
	ErrChk(hipEventRecord(start,0));

	for (int run = 0; run<N_RUNS; ++run){
		gemm_256<<<grid_size, block_size, sizeof(float)*4*128*8>>>(dev_M, dev_N, dev_K, dev_A, dev_B, dev_C, dev_T, dev_Ba);
		KernelErrChk();
	}

	ErrChk(hipEventCreate(&stop));
	ErrChk(hipEventRecord(stop,0));
	ErrChk(hipEventSynchronize(stop));
	ErrChk(hipEventElapsedTime(&elapsedTime, start,stop));

	time = elapsedTime/N_RUNS;
	time /= 1.0e3; //convert time unit from millisecond to second
	gflops_per_sec   = gflops / time;
	printf("%f\n", gflops_per_sec);

	for (int i=0; i<BATCH; ++i){
		ErrChk(hipFree(A[i]));		
		ErrChk(hipFree(B[i]));		
		ErrChk(hipFree(C[i]));		
	}

	free(M);
	free(N);
	free(K);
	free(A);
	free(B);
	free(C);
	free(t_strategy);

	ErrChk(hipFree(dev_M));		
	ErrChk(hipFree(dev_N));		
	ErrChk(hipFree(dev_K));		
	ErrChk(hipFree(dev_T));		

	ErrChk(hipFree(dev_A));		
	ErrChk(hipFree(dev_B));		
	ErrChk(hipFree(dev_C));		

	return 0;
}
